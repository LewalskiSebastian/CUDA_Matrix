#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include "iostream"
#include <ctime>

#define liczba_blokow 16
#define liczba_watkow 512
#define iloczyn 8192

hipError_t addWithCuda(unsigned long long liczba, bool czy_niepierwsza_cuda, unsigned long long pierwiastek);

__global__ void is_prime_Kernel(unsigned long long liczba, bool &czy_niepierwsza, unsigned long long pierwiastek)
{
	unsigned long long idx = threadIdx.x + 2 + blockIdx.x * blockDim.x;
	while (idx <= pierwiastek) {
		if (liczba % idx == 0) czy_niepierwsza = true;
		idx = idx + iloczyn;
	}
}

int main()
{
	bool czy_niepierwsza = false;
	unsigned long long liczba;
	unsigned long long pierwiastek;

	bool czy_niepierwsza_cuda = false;

	printf("Podaj liczbe: \n");
	//scanf("%llu", &potential_prime);
	liczba = 100000150499;
	printf("Twoja liczba to: %llu\n", liczba);
	pierwiastek = sqrt(liczba);

	std::cout << "(CUDA): ";

	clock_t start = clock();
	if (pierwiastek * pierwiastek == liczba) {
		std::cout << "Liczba " << liczba << " nie jest liczba pierwsza poniewaz dzieli sie przez pierwiastek " << pierwiastek << "\n";
		printf("Czas wykonywania: %lu ms\n", clock() - start);
		czy_niepierwsza = true;
	}
	int dzielnik = 1;
	if (!czy_niepierwsza) {
		do {
			dzielnik++;
		} while (liczba % dzielnik != 0 && dzielnik < pierwiastek + 1);
		if (dzielnik == pierwiastek + 1) {
			std::cout << "Liczba " << liczba << " jest liczba pierwsza \n";
			printf("Czas wykonywania: %lu ms\n", clock() - start);
		}
		else {
			std::cout << "Liczba " << liczba << " nie jest liczba pierwsza, poniewaz dzieli sie przez " << dzielnik << "\n";
			printf("Czas wykonywania: %lu ms\n", clock() - start);
		}
	}

	std::cout << "(CUDA): ";

	hipError_t cudaStatus = addWithCuda(liczba, czy_niepierwsza_cuda, pierwiastek);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	return 0;
}

hipError_t addWithCuda(unsigned long long liczba, bool czy_niepierwsza_cuda, unsigned long long pierwiastek)
{
	bool* czy_niepier;
	float computing_time = 0;
	hipError_t cudaStatus;
	hipEvent_t start_counting, stop_counting;

	cudaStatus = hipEventCreate(&start_counting);
	cudaStatus = hipEventCreate(&stop_counting);

	hipEventRecord(start_counting, 0);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&czy_niepier, sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	is_prime_Kernel << < liczba_blokow, liczba_watkow >> > (liczba, *czy_niepier, pierwiastek);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(&czy_niepierwsza_cuda, czy_niepier, sizeof(bool), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	hipEventRecord(stop_counting, 0);
	hipEventSynchronize(stop_counting);

	hipEventElapsedTime(&computing_time, start_counting, stop_counting);

	hipEventDestroy(start_counting);
	hipEventDestroy(stop_counting);

	if (czy_niepierwsza_cuda)
	{
		std::cout << "Liczba " << liczba << " nie jest liczba pierwsza \n";
	}
	else
	{
		std::cout << "Liczba " << liczba << " jest liczba pierwsza \n";
	}
	printf("Czas wykonywania: %lu ms\n", computing_time);

Error:
	hipFree(czy_niepier);
	return cudaStatus;
}